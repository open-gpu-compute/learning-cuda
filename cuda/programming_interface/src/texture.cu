#include "hip/hip_runtime.h"
/*
Compiling with nvcc:
nvcc texture.cu -o texture -std=c++11
./texture
*/
// Iniating a cuda texture object
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
using namespace std::chrono;
using namespace std;
// Define CUDA texture Desciption
struct hipTextureDesc
{
    enum hipTextureAddressMode addressMode[3];
    enum hipTextureFilterMode  filterMode;
    enum hipTextureReadMode    readMode;
    int                         sRGB;
    int                         normalizedCoords;
    unsigned int                maxAnisotropy;
    enum hipTextureFilterMode  mipmapFilterMode;
    float                       mipmapLevelBias;
    float                       minMipmapLevelClamp;
    float                       maxMipmapLevelClamp;
};
#define N 1024

// texture object is a kernel argument
__global__ void kernel(hipTextureObject_t tex) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float x = tex1Dfetch<float>(tex, i);
  //  Do something with x
}


int main() {
  // declare and allocate memory
  float *buffer;
  hipMalloc(&buffer, N*sizeof(float));

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = buffer;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  
  resDesc.res.linear.sizeInBytes = N*sizeof(float);

  const hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  kernel <<<512, 512>>>(tex); // pass texture as argument

  // destroy texture object
  hipDestroyTextureObject(tex);

  hipFree(buffer);
}