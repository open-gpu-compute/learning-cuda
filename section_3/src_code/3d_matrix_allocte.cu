/*
Compiling with nvcc:
nvcc 3d_matrix_allocte.cu -o 3d_matrix_allocte -std=c++11
./3d_matrix_allocte
Sample Output:
Matrix Allocated
Time taken for matrix allocation : 9 microseconds
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;
// The following code sample allocates a width x height x 
// depth 3D array of floating-point values and shows how 
// to loop over the array elements in device code
// Device code
__global__ void matrixLoop(hipPitchedPtr devPitchedPtr,
                         int width, int height, int depth)
{
    char* deice_pointer = (char* )devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * height;
    for (int z = 0; z < depth; ++z) {
        char* slice = deice_pointer + z * slicePitch;
        for (int y = 0; y < height; ++y) {
            float* row = (float*)(slice + y * pitch);
            for (int x = 0; x < width; ++x) {
                float element = row[x];
            }
        }
    }
}
// Host code
int main()
{
    int width = 128, height = 128, depth = 128;
    //cuda data structure used to dimension o the matrix
    hipExtent extent = make_hipExtent(width * sizeof(float),
                                        height, depth);
    // declare cuda pitched memory pointer                                    
    hipPitchedPtr devPitchedPtr;
    // allocate 3d matrix
    hipMalloc3D(&devPitchedPtr, extent);
    printf("Matrix Allocated\n");
    matrixLoop<<<100, 512>>>(devPitchedPtr, width, height, depth);
}